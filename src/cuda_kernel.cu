#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <iostream>
#include "param_struct.hpp"
//ERROR HANDLING FOR GPU CALLS (TAKEN FROM STACK OVERFLOW)
#define gpuErrchk(ans) {gpuAssert((ans),  __FILE__, __LINE__);}
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
	if(code!=hipSuccess){
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if(abort) exit(code);
	}
}
using namespace std;
//THIS IS JUST FOR EXPLORATION, WILL BE DELETED
__global__ void test_kernel(int size, double * array_x, double * array_y){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i<size)
		for(int i=0;i<size;i++)
			array_y[i]=array_y[i]+array_x[i];
}
//GEMM KERNEL, NO OPTIMIZATIONS, MAY INCLUDE TILING ETC
__global__ void gemm_kernel(const double *A, const double *B, double *C, const int n, const int k, const int m){
	int col =  blockIdx.x*blockDim.x + threadIdx.x;
	int row =  blockIdx.y*blockDim.y + threadIdx.y;
	if(row<n && col<m)
		for(int p=0; p<k; p++)
			C[row*m+col] = C[row*m+col] + A[row*k+p]*B[p*k+col];

}
__global__ void jacobi_kernel(double *U, double *Unew, double *A, double *F, int N){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	//for(int i=0;i<N;i++){
	double sigma = 0.0;

	if(i<N && y==0){
		for(int j=0; j<N; j++){
			if(j!=i)
				sigma+=A[j+i*N]*U[j];
		}
		Unew[i] = (F[i]-sigma)/A[i+i*N];
	}
	//}
}
//JACOBI SMOOTHING EXECUTED ON HOST/DEVICE TO MEASURE ERRORS. CALLED WITH KERNEL=TRUE FOR GPU LAUNCH
double jacobi_smoother(double *U, double *Unew, double *A, double *F, int N, dim3 blocksPerGrid, dim3 threadsPerBlock, bool KERNEL){
	int MAX_ITER = 1000;
	int iter=0;
	double *temp;
	//runs through jacobi naively, swaps pointers, iterates
	while(iter<MAX_ITER){
		if(KERNEL){
			//printf("%d %d\n", U, Unew);
			jacobi_kernel<<<blocksPerGrid, threadsPerBlock>>>(U, Unew, A, F, N);
		}else{
			printf("%d %d\n", U, Unew);
			
			for(int i=0;i<N;i++){
				double sigma = 0.0;
				for(int j=0; j<N; j++){
					if(j!=i)
						sigma+=A[j+i*N]*U[j];
				}
				Unew[i]= (F[i]-sigma)/A[i+i*N];
			}

		}
		temp = U;
		U = Unew;
		Unew = temp;
		iter++;
	}
	if(KERNEL)
		return 0.0;
	double jacobi_error =0.0;
	for(int i=0;i<N;i++){
		jacobi_error+=U[i];
		//printf("%lf ", U[i]);
	}
	//printf("\n");
	return jacobi_error;
}

//PSO will call kernel_wrapper with different parameters and kernel_wrapper will evaluate kernel and store statistics
void kernel_wrapper(int iteration, dim3 blocksPerGrid, dim3 threadsPerBlock, record_t * records){

	//ARRAY OF STRUCTURES OR STRUCTURE OF ARRAYS? ARRAY OF STRUCTURES SEEMS TO MAKE MORE SENSE
	//CLEANER CODE AND THE WHOLE STRUCTURE WILL BE ACCESSED SEQUENTIALLY, NOT AN INTERNAL ARRAY.
	records[iteration].parameters.threads_per_block = threadsPerBlock;
	records[iteration].parameters.blocks_per_grid = blocksPerGrid;	

	//INITIALIZE GEMM MATRICIES
	size_t n, k, m;
	n = k = m = 1024;
	double *A, *B, *C;
	A = (double *)malloc(n*k*sizeof(double)), B= (double *)malloc(k*m*sizeof(double)), C=(double *)malloc(n*m*sizeof(double));
	for(int i=0;i<n;i++)
		for(int j=0;j<k;j++)
			A[i*k+j]=2.0*i+1.0*j+1.0;
	for(int i=0;i<k;i++)
		for(int j=0;j<m;j++)
			B[i*m+j]=2.0*i+1.0*j+1.0;
	for(int i=0;i<n;i++)
		for(int j=0;j<m;j++)
			C[i*m+j]=0.0;
	
	double gemm_checksum = 0.0;
	for(int i=0;i<n;i++)
		for(int p=0;p<k;p++)
			for(int j=0;j<m;j++)
				gemm_checksum+=A[i*k+p]*B[p*k+j];
	
	//DECLARE DEVICE POINTERS, CUDAMALLOC,  AND COPY MEMORY
	double *A_d, *B_d, *C_d;
	gpuErrchk(hipMalloc(&A_d, n*k*sizeof(double)));
	gpuErrchk(hipMemcpy(A_d, A, n*k*sizeof(double), hipMemcpyHostToDevice)); 
	gpuErrchk(hipMalloc(&B_d, k*m*sizeof(double)));
	gpuErrchk(hipMemcpy(B_d, B, k*m*sizeof(double), hipMemcpyHostToDevice)); 
	gpuErrchk(hipMalloc(&C_d, n*m*sizeof(double)));
	gpuErrchk(hipMemcpy(C_d, C, n*m*sizeof(double), hipMemcpyHostToDevice)); 
	
	//LAUNCH KERNEL, RECORD TIME, COPY KERNEL RESULTS TO HOST
	hipEvent_t gemm_start, gemm_stop;
	gpuErrchk(hipEventCreate(&gemm_start));
	gpuErrchk(hipEventCreate(&gemm_stop));
	gpuErrchk(hipEventRecord(gemm_start));
	gemm_kernel<<<blocksPerGrid,threadsPerBlock>>>(A_d, B_d, C_d, n, k, m);
	gpuErrchk(hipEventRecord(gemm_stop));
	gpuErrchk(hipMemcpy(C, C_d, n*m*sizeof(double), hipMemcpyDeviceToHost));
	gpuErrchk(hipEventSynchronize(gemm_stop));
	
	//CALCULATE ERROR
	for(int i =0;i<n;i++)
		for(int j=0; j<m; j++)
			gemm_checksum-=C[i*m+j];
	
	cout<<"GEMM\n"<<"Error "<<gemm_checksum<<endl;
	gpuErrchk(hipEventElapsedTime(&records[iteration].gemm_time, gemm_start, gemm_stop));
	cout<<"Time "<< records[iteration].gemm_time/1e3<< " Seconds"<<endl;
	hipFree(A_d);
	hipFree(B_d);
	hipFree(C_d);
	free(A), free(B), free(C);

	//SYNCHRONIZE BETWEEN KENRNELS? WILL IMPLEMENT MULTIPLE STREAMS
	//gpuErrchk(hipDeviceSynchronize());

	//JACOBI
	//INITIALIZE VECTORS U AND F,  MATRIX J , JxU=F (nxm)x(mx1)=(nx1)
	double *U, *F, *Unew,*U_d, *F_d, *Unew_d, *J_d;
	U = (double *)malloc(m*sizeof(double));
	Unew = (double *)malloc(m*sizeof(double));
	F = (double *)malloc(n*sizeof(double));
	for(int i=0; i<m; i++){
		U[i]=0.0;
		Unew[i]=0.0;
		F[i]=1.0;
	}
	double *J=(double *)malloc(n*m*sizeof(double));
	for(int i=0;i<n;i++){
		for(int j=0;j<n;j++){
			if(j==i-1 || j==i+1)
				J[i*n+j]=-1.0;
			else if(j==i)
				J[i*n+j]=2.0;
			else
				J[i*n+j]=0.0;
		}
	}
	//HOST SOLUTION
	double jacobi_checksum = jacobi_smoother(U, Unew, J, F, m, blocksPerGrid, threadsPerBlock, false);
	
	gpuErrchk(hipMalloc(&U_d, m*sizeof(double)));
	gpuErrchk(hipMalloc(&Unew_d, m*sizeof(double)));
	gpuErrchk(hipMemcpy(U_d, U, m*sizeof(double), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(Unew_d, Unew, m*sizeof(double), hipMemcpyHostToDevice));
	gpuErrchk(hipMalloc(&F_d, n*sizeof(double)));
	gpuErrchk(hipMemcpy(F_d, F, n*sizeof(double), hipMemcpyHostToDevice));
	gpuErrchk(hipMalloc(&J_d, n*m*sizeof(double)));
	hipMemcpy(J_d, J, n*m*sizeof(double), hipMemcpyHostToDevice); 

	//KERNEL, TIME, MEMCPY
	hipEvent_t jacobi_start, jacobi_stop;
	gpuErrchk(hipEventCreate(&jacobi_start));
	gpuErrchk(hipEventCreate(&jacobi_stop));
	gpuErrchk(hipEventRecord(jacobi_start));
	//TRIED DEBUGING BY EXPLICITLY SWAPPING POINTERS COMMENT jacobi_smoother() BELOW TO DEBUG
	/*for(int k=0;k<1000; k++){
		if(k%2==0)
			jacobi_kernel<<<blocksPerGrid, threadsPerBlock>>>(U_d, Unew_d, J_d, F_d, m);
		else
			jacobi_kernel<<<blocksPerGrid, threadsPerBlock>>>(Unew_d, U_d, J_d, F_d, m);
	}*/
	jacobi_smoother(U_d, Unew_d, J_d, F_d, m, blocksPerGrid, threadsPerBlock, true);
	gpuErrchk(hipEventRecord(jacobi_stop));
	gpuErrchk(hipMemcpy(U, U_d, m*sizeof(double), hipMemcpyDeviceToHost));
	gpuErrchk(hipEventSynchronize(jacobi_stop));
	
	//UPDATE CHECKSUM WITH GPU SOLUTION
	for(int i=0;i<m;i++){
		printf("%lf ", U[i]);
		jacobi_checksum-=U[i];
	}
	printf("\n");
	cout<<"JACOBI\n"<<"Error "<<jacobi_checksum<<endl;
	gpuErrchk(hipEventElapsedTime(&records[iteration].jacobi_time, jacobi_start, jacobi_stop));
	cout<<"Time "<< records[iteration].jacobi_time/1e3<< " Seconds"<<endl;
	

	hipFree(U_d), hipFree(Unew_d), hipFree(F_d), hipFree(J_d);
	free(U), free(Unew), free(F), free(J);
}

