#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <iostream>
#include "param_struct.hpp"

using namespace std;
__global__ void test_kernel(int size, double * array_x, double * array_y){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i<size)
		for(int i=0;i<size;i++)
			array_y[i]=array_y[i]+array_x[i];
}
__global__ void gemm_kernel(const double *A, const double *B, double *C, const int n, const int k, const int m){
	int col =  blockIdx.x*blockDim.x + threadIdx.x;
	int row =  blockIdx.y*blockDim.y + threadIdx.y;
	if(row<n && col<m)
		for(int p=0; p<k; p++)
			C[row*m+col] = C[row*m+col] + A[row*k+p]*B[p*k+col];

}


//PSO will call kernel_wrapper with different parameters and kernel_wrapper will evaluate kernel and store statistics
void kernel_wrapper(int iteration, dim3 blocksPerGrid, dim3 threadsPerBlock, record_t * records){

	//ARRAY OF STRUCTURES OR STRUCTURE OF ARRAYS? ARRAY OF STRUCTURES SEEMS TO MAKE MORE SENSE
	//CLEANER CODE AND THE WHOLE STRUCTURE WILL BE ACCESSED SEQUENTIALLY, NOT AN INTERNAL ARRAY.
	records[iteration].parameters.threads_per_block = threadsPerBlock;
	records[iteration].parameters.blocks_per_grid = blocksPerGrid;	

	//INITIALIZE GEMM MATRICIES
	size_t n, k, m;
	n = k = m = 32;
	double *A, *B, *C;
	A = (double *)malloc(n*k*sizeof(double)), B= (double *)malloc(k*m*sizeof(double)), C=(double *)malloc(n*m*sizeof(double));
	for(int i=0;i<n;i++)
		for(int j=0;j<k;j++)
			A[i*k+j]=2.0*i+1.0*j;
	for(int i=0;i<k;i++)
		for(int j=0;j<m;j++)
			B[i*m+j]=2.0*i+1.0*j;
	for(int i=0;i<n;i++)
		for(int j=0;j<m;j++)
			C[i*m+j]=0.0;
	double checksum = 0.0;
	for(int i=0;i<n;i++)
		for(int p=0;p<k;p++)
			for(int j=0;j<m;j++)
				checksum+=A[i*k+p]*B[p*k+j];
	
	//DECLARE DEVICE POINTERS, CUDAMALLOC,  AND COPY MEMORY
	double *A_d, *B_d, *C_d;
	hipMalloc(&A_d, n*k*sizeof(double));
	hipMemcpy(A_d, A, n*k*sizeof(double), hipMemcpyHostToDevice); 
	hipMalloc(&B_d, k*m*sizeof(double));
	hipMemcpy(B_d, B, k*m*sizeof(double), hipMemcpyHostToDevice); 
	hipMalloc(&C_d, n*m*sizeof(double));
	hipMemcpy(C_d, C, n*m*sizeof(double), hipMemcpyHostToDevice); 
	//LAUNCH KERNEL, RECORD TIME, COPY KERNEL RESULTS TO HOST
	hipEvent_t gemm_start, gemm_stop;
	hipEventCreate(&gemm_start), hipEventCreate(&gemm_stop);
	hipEventRecord(gemm_start,0);
	gemm_kernel<<<blocksPerGrid,threadsPerBlock>>>(A_d, B_d, C_d, n, k, m);
	hipEventRecord(gemm_stop,0);
	hipMemcpy(C, C_d, n*m*sizeof(double), hipMemcpyDeviceToHost);
	hipEventSynchronize(gemm_stop);
	
	double sum = 0.0;
	for(int i =0;i<n;i++)
		for(int j=0; j<m; j++)
			sum+=C[i*m+j];
	
	cout<<"GEMM\n"<<"Error "<<checksum-sum<<endl;
	hipEventElapsedTime(&records[iteration].gemm, gemm_start, gemm_stop);
	cout<<"time "<< records[iteration].gemm/1e3<< "seconds"<<endl;
	hipFree(A_d);
	hipFree(B_d);
	hipFree(C_d);
	free(A), free(B), free(C);

}



